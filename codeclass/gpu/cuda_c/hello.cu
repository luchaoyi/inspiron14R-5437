
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void HelloGpu() {
	__shared__ int arr[10];
	for(int i = 0; i < 10; i++) {
		printf("%d ", arr[i]);
	}
	printf("\nHello GPU\n");
}

int main() {
	printf("Hello CPU\n");
	HelloGpu<<<1,10>>>();
	hipDeviceReset();
	return 0;
}
