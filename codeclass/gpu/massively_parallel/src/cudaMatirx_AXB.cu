#include "hip/hip_runtime.h"
//使用二维Grid将矩阵分成一个个block方块
//block方块是二维block,每一个线程负责计算一个元素
//C[i][j]= 对k循环求和 A[i][k]*B[k][j]


//Width * Width 方阵
//Idx Index


__global__ void MatrixMulKernel(float * d_M,float * d_N,float * d_P,int Width)
{
	//blockIdx内置dim3变量，每个线程有自己的blockIdx和threadIdx用于定位
	int Col=blockIdx.x*blockDim.x+threadIdx.x;
	int Row=blockIdx.y*blockDim.y+threadIdx.y;

	if(Row<Width) && (Col<Width) {
		float Pvalue=0;
		for(intk=0;intk<Width;intk++){
			Pvalue=d_M[Row*Width+k]*d_N[k*Width+Col];
		}
		d_P[Row*Width+Col] = Pvalue;
	}
}


//矩阵以行优先(连续)或列优先(连续)存储
//A X B 时 A的一行*B的一列，必定有一个不满足连续访存
//因此对另一个进行转置让其满足矩阵乘法


//使用共享存储器加载分块优化全局访问的矩阵乘法
#define TILE_WIDTH 16
__global__ void MatrixMulKernel(float * d_M,float * d_N,float * d_P,int Width){
	__shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
	__shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

	int bx=blockIdx.x;int by=blockIdx.y;
	int tx=threadIdx.x;int ty=threadIdx.y;

	//一个block共享 __shared__,因此一个block负责一个块,块大小为TILE_WIDTH
	int Row=by*TILE_WIDTH+ty;
	int Col=bx*TILE_WIDTH+tx;

	float Pvalue=0;
	for(int m=0;m<Width/TILE_WIDTH;++m){
		//一块一块加载元素，一块一块计算
		Mds[ty][tx]=d_M[Row*Width+m*TILE_WIDTH+tx];
		Nds[ty][tx]=d_N[(m*TILE_WIDTH+ty)*Width+Col];
		__syncthreads();

		for(int k=0;k<TILE_WIDTH;++k){
			Pvalue+=Mds[ty][k]*Nds[k][tx];
		}
		__syncthreads();
	}

	d_P[Row*Width+Col]=Pvalue;
}
