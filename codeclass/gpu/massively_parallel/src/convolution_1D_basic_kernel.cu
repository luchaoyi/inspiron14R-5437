#include "hip/hip_runtime.h"
//N is input
//M is卷积模板
//P is output

__global__ void convolution_1D_basic_kernel(float *N,float *M,float *P,int Mask_width,int width){

	//线程内使用寄存器存储变量
	//避免多次访问全局存储器
	int i=blockIdx.x*blockDim.x+threadIdx.x;

	float Pvalue=0;
	int N_start_point=i-(Mask_width/2);

	for(j=0;j<Mask_width;j++){
		if(N_start_point+j>=0 && N_start_point+j<=Width){
			Pvalue+=N[N_start_point+j]*M[j];
		}
	}

	P[i]=Pvalue;

}

//=>优化

//卷积核M不改变，且一般相对于N很小
//使用常数存储器存储

#define MAX_MASK_WIDTH 10
__constant__ float M[MAX_MASK_WIDTH];
hipMemcpyToSymbol(HIP_SYMBOL(M),h_M,Mask_width*sizeof(float));  //主机h_M->M 复制数据到常数存储器

//代码没有改动只是使用常数存储器存储了M 
//常数存储器速度不比全局存储器快，但它存储的值不变，因此GPU可以将它直接加载到高速缓存中(高速缓存是透明的,程序不操作它),不会出现不一致问题

__global__ void convolution_1D_basic_kernel(float *N,float *P,int Mask_width,int width){

	//线程内使用寄存器存储变量
	//避免多次访问全局存储器
	int i=blockIdx.x*blockDim.x+threadIdx.x;

	float Pvalue=0;
	int N_start_point=i-(Mask_width/2);

	for(j=0;j<Mask_width;j++){
		if(N_start_point+j>=0 && N_start_point+j<=Width){
			Pvalue+=N[N_start_point+j]*M[j];
		}
	}

	P[i]=Pvalue;

}



//分块卷积，每个block负责一个分块，每个thread负责分块内的每个元素
//共享存储器是block共享，不同block私有,使用共享存储器存储每个block负责的分段的元素，减少直接访问全局存储N

//cuda编程与host编程不同，会经常声明变量存储中间结果为为了使用寄存器加快速度.

#define TILE_SIZE 分块大小

__global__ void convolution_1D_basic_kernel(float *N,float *P,int Mask_width,int width){
	int i=blockIdx.x*blockDim.x+threadIdx.x;  //一维block,每个block内是一维thread 


	__shared__ float N_ds[TILE_SIZE]; //共享存储器是block私有的
	N_ds[threadIdx.x]=N[i]; //将每段元素先载入到共享存储中,每个线程载入一个

	__syncthreads(); //同步等待block内线程都已载入元素

	int This_tile_start_point = blockIdx.x*blockDim.x;
	int Next_tile_start_point=This_tile_start_point+blockDim.x;//next=(this+1)*len
	int N_start_point=i-(Mask_width/2); //此线程计算元素，卷积开始位置
	float Pvalue=0;

	
	for(int j=0;j<Mask_width;j++){
		int N_index=N_start_point+j;
		if(N_index>=0&&N_index<width){ //在此区域外补0,不算
			if(
					(N_index>=This_tile_start_point) 
					&&
					(N_index<Next_tile_start_point)
					)//元素在此段内，则存储在共享存储器中
				Pvalue+=N_ds[threadIdx.x+j-(Mask_width/2)]*M[j];
			else

			/*书中提到的复杂的分块卷积，是将这些光环元素，即对于此分块的边界元素也加载到了共享存储器中,这样需要更多的控制逻辑.
			这里利用了一个事实是N元素可能已经被加载到了高速缓存,因此并不会频繁从N加载
			*/
				Pvalue+=N[N_index]*M[j];  //没在则从N中加载
		}
	}
}
