#include<hip/hip_runtime.h>
#include<stdio.h>


//device is SPMD so all线程执行同一份代码
//调用kernel函数会在device生成线程网格，一个网格是包含多个线程的线程块
//每个线程块内的线程由threadIdx标识
//每一个线程块由blockIdx标识,块内线程数blockDim
__global__ void vecAddKernel(float *A,float *B,float *C,int n){
	//因此每一个线程计算的i是唯一的
	int i=blockIdx.x*blockDim.x+threadIdx.x; 
	//可以以此i当作下标
	if(i<n) C[i]=A[i]+B[i];
}



//使用二维网格处理二维图片
__global__ void PictureKernel(float *d_Pin,float *d_Pout,int n,int m){
	int Row=blockIdx.y*blockDim.y+threadIdx.y;
	int Col=blockIdx.x*blockDim.x+threadIdx.x;
	if((Row<m)&&(Col<n)){
		d_Pout[Row*n+Col]=2*d_Pin[Row*n+Col];
	}

}




__host__ void vecAdd(float *A,float *B,float *C,int n) //激活__global__函数的__host__函数，称为stub函数
{
	int size=n*sizeof(float);
	float *d_A,*d_B,*d_C;


	hipMalloc((void **)&d_A,size);
	//destination,source,bytes?,->?
	hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
	hipMalloc((void **)&d_B,size);
	hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);
	hipMalloc((void **)&d_C,size);

	//激活kernel函数,在device执行向量运算
	//<<<block的number,block的线程数>>>



	dim3 dimGrid(ceil(n/256.0),1,1);
	dim3 dimBlock(256,1,1);
	vecAddKernel<<<dimGrid,dimBlock>>>(d_A,d_B,d_C,size);
	//vecAddKernel<<<ceil(n/256.0),256>>>(d_A,d_B,d_C,size);
	//回收result
	//d_C->C
	hipDeviceSynchronize();
	hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);
	//Free device
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

}

#define N 1000

int main(int argc,char** argv){

	float A[N];
	float B[N];
	float C[N];
	int i;


	for(i=0;i<N;i++){
		A[i]=1.0;
		B[i]=2.0;
		C[i]=0.0;
	}
	

	vecAdd(A,B,C,N);
	for(i=0;i<N;i++){
		printf("%f\n",C[i]);
	}
	
	return 0;
}



