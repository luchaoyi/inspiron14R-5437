#include "hip/hip_runtime.h"
//串行
//A*x+y 矩阵×向量+向量

//A=
//row_ptr
//data
//col_index

void SPMV_CSR()
{

for (int row=0;row<num_rows;row++){
	float dot=0;
	int row_start=row_ptr[row];
	int row_end=row_ptr[row+1];

	for(int elem=row_start;elem<row_end;elem++){
		dot+=data[elem]*x[col_index[elem]];

	}
	y[row]+=dot;
}
}

//并行化
//每一行dot是独立的 A[i]*x+y独立


__global__ void SPMV_CSR(int num_rows,float *data,int *col_index,
		int *row_ptr,float *x,float *y
		){

	int row=blockIdx.x*blockDim.x+threadIdx.x;

	if(row<num_rows){
		float dot=0;
		int row_start=row_ptr[row];
		int row_end=row_ptr[row+1];

		for(int elem=row_start;elem<row_end;elem++){
			dot+=data[elem]*x[col_index[elem]];

		}
		y[row]+=dot;
	}
}




//填充与转置
//行长短不一样，则wrap内线程迭代次数不一致,可以填充元素使行一样长消除控制流分支
//合并访问
//每一个线程访问A的不同行执行dot运算
//若按列存储则不同行同一列元素相邻存储，则相邻线程访问相邻元素可以合并访问,优化速度

//排序划分
//为使wrap尽量对齐，可以按行元素数量对矩阵排序，将元素数量接近的划分到一个段，为每一个段构造一个kernel或在一个block内处理
//在如线性方程求解等问题上，行互换是不影响结果的
//在影响结构的问题上需要记录行原来所属位置


