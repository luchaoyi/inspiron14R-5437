#include "hip/hip_runtime.h"
//串行scan
//双精度的sum可以避免累加误差
void scan(float *x,float *y,int Max_i){
	double sum;
	sum=x[0];
	for(i=1;i<Max_i;i++){
		sum+=x[i];
		y[i]=sum;
	}
}




//跳步scan
//开始累加相邻元素，
//然后累加相隔2元素
//...stride*=2

__global__  stride_scan(float *X,float *Y,int InputSize){
	__shared__ float XY[SECTION_SIZE];//每一个block内的scan
	int i=blockIdx.x+blockDim.x+threadIdx.x;
	if(i<InputSize){
		XY[threadIdx.x]=X[i];
	}
	
	//scan on XY

	for(unsigned int stride=1;stride<=threadIdx.x;stride*=2){
		__syncthreads();
		XY[threadIdx.x]+=XY[threadIdx.x-stride];
	}
	Y[i]=XY[i];
}


//归约树
//先归约，在倒过来分发
__global__  tree_reduce_scan(float *X,float *Y,int InputSize){

	int i=blockIdx.x+blockDim.x+threadIdx.x;
	if(i<InputSize){
		XY[threadIdx.x]=X[i];
	}

	for(unsigned int stride=1;stride<=blockDim.x;stride*=2){
		__syncthreads();
		int index=(threadIdx.x+1)*2*stride-1;

		if(index<blockDim.x){
			XY[index]+=XY[index-stride];
		}

	for(int stride=SECTION_SIZE/4;stride>0;stride/=2){
		__syncthreads();
		int index=(threadIdx.x+1)*stride*2-1;
		if(index+stride<BLOCK_SIZE)
		{
			XY[index+stride]++XY[index];
		}
	}

	__syncthreads();
	Y[i]=XY[threadIdx.x];
}
}

//任意长度的scan，可以分段采用上述方法scan
//然后将每段最后一个元素scan
//然后每段加上scan后的数组







