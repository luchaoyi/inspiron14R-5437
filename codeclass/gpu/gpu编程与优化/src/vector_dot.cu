#include "hip/hip_runtime.h"
//c=A.B
//求和累加

//不同线程计算乘积,然后在求和归约
__global__ void vdp_1(DATATYPE *a,DATATYPE *b,DATATYPE *c,int n){
	__shared__ DATATYPE tmp[threadnum];

	const int tidx=threadIdx.x;
	const int t_n=blockDim.x;

	int tid=tidx;
	double tmp=0.0;

	//t_n不一定等于n,一个线程可能计算多个元素
	while(tid<n){
		temp+=a[tid]*b[tid];
		tid+=t_n;
	}
	tmp[tidx]=temp;
	__syncthreads();


	int i=2,j=1;

	while(i<threadnum){
		if(tidx%i==0){
			tmp[tidx]+=tmp[tid+j];
		}
		__syncthreads();
		i*=2;
		j*=2;
	}


	if(tidx==0){
		c[0]=tmp[0];
	}

//vdp_1<<<1,threadnum>>>(...) 单block
}

//可以使用多个block 
//可以GPU初步归约后
//发送到CPU，CPU再次归约
